#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <omp.h>
#define BLOCK_SIZE 16

__global__ void gpu_mult_matrix(int *a,int *b, int *c, int m, int n, int k)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < k && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
} 

__global__ void gpu_square_mul_matrix(int *d_a, int *d_b, int *d_result, int n) 
{
    __shared__ int  _a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int  _b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by BLOCK_SIZE
             _a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
             _a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if(idx >= n*n)
        {
             _b[threadIdx.y][threadIdx.x] = 0;
        }  
        else
        {
             _b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp +=  _a[threadIdx.y][k] *  _b[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
}


__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols) 
{
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows) 
    {
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;
        mat_out[trans_pos] = mat_in[pos];
    }
}


void cpu_mul_matrix(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) 
    {
        for (int j = 0; j < k; ++j) 
        {
            int tmp = 0.0;
            for (int h = 0; h < n; ++h) 
            {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

int dtn(int n, int min_n) 
{
    int max_tn = n / min_n;
    const int g_ncore = omp_get_num_procs();
    int tn = max_tn > g_ncore ? g_ncore : max_tn; 
    if(tn < 1)
    {
        tn = 1; 
    }
    return tn; 
}


/void omp_mm(int *a, int row_a, int col_a, int *b, int row_b,int col_b, int *c)
{
    if ( col_a != row_b ) 
    {
        return; 
    }
    int i, j, k;
    int index;
    int border = row_a * col_b;
    i = 0;
    j = 0;

    #pragma omp parallel for private(i,j,k) num_threads(dtn(border, 1))
    for ( index = 0; index < border; index++ ) 
    {
        i = index / col_b; j = index % col_b;
        int row_i = i * col_a; 
        int row_c = i * col_b;
        c[row_c+j] = 0;
        for ( k = 0; k < row_b; k++ ) 
        {
            c[row_c + j] += a[row_i+k] * b[k*col_b+j]; 
        }
    } 
}

int main(int argc, char const *argv[])
{
    int m, n, k;
    srand(2222);
    printf("please type in m n and k\n");
    scanf("%d %d %d", &m, &n, &k);

    // allocate memory in host RAM, h_cc is used to store CPU result
    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*m*n);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*k);
    hipHostMalloc((void **) &h_c, sizeof(int)*m*k);
    hipHostMalloc((void **) &h_cc, sizeof(int)*m*k);

    // random initialize matrix A
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
        }
    }

    // random initialize matrix B
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 1024;
        }
    }

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

    
    hipEventCreate(&start);
    hipEventCreate(&stop);

    
    hipEventRecord(start, 0);
   
    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*m*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*k);
    hipMalloc((void **) &d_c, sizeof(int)*m*k);

   
    hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // Launch kernel 
    if(m == n && n == k)
    {
        gpu_square_mul_matrix<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);    
    }
    else
    {
        gpu_mul_matrix<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);    
    }
    
 hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);

    // start the CPU version
    hipEventRecord(start, 0);

      omp_mm(h_a, m, n, h_b, n, k, h_cc);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on CPU: %f ms.\n\n", m, n, n, k, cpu_elapsed_time_ms);

    // validate results computed by GPU
    int all_ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            //printf("[%d][%d]:%d == [%d][%d]:%d, ", i, j, h_c[i*k + j], i, j, h_c[i*k + j]);
            if(h_c[i*k + j] != h_c[i*k + j])
            {
                all_ok = 0;
            }
        }
        //printf("\n");
    }


    if(all_ok)
    {
        printf("all results are correct!!!, speedup = %f\n", cpu_elapsed_time_ms / gpu_elapsed_time_ms);
    }
    else
    {
        printf("incorrect results\n");
    }

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);
    return 0;
}